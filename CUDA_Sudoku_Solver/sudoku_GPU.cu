#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "stdio.h"
#include "stdlib.h"
#include "sudoku_GPU.cuh"


#define DIMENSION 9
#define SUBSQUARE_DIMENSION 3
#define BLOCK_DIMENSION 512
#define GRID_DIMENSION 1024
#define MAX_MEMORY_USAGE 0.99999
__device__
int row_used_numbers(char* sudokus_arr, unsigned long long row, int* masks) {
	int used_numbers = 0;
	for (int i = 0; i < DIMENSION; i++) {
		used_numbers = used_numbers | masks[sudokus_arr[i + row]];
	}
	return used_numbers;
}

__device__
int column_used_numbers(char* sudokus_arr, unsigned long long column, int* masks) {
	int used_numbers = 0;
	for (int i = 0; i < DIMENSION; i++) {
		used_numbers = used_numbers | masks[sudokus_arr[column + i * DIMENSION]];
	}
	return used_numbers;
}

__device__
int subsquare_used_numbers(char* sudokus_arr, unsigned long long subsquare_top_left_cell, int* masks) {
	int used_numbers = 0;
	for (int i = 0; i < SUBSQUARE_DIMENSION; i++) {
		for (int j = 0; j < SUBSQUARE_DIMENSION; j++) {
			used_numbers = used_numbers | masks[sudokus_arr[i + subsquare_top_left_cell + j * DIMENSION]];
		}
	}
	return used_numbers;
}

__device__
int get_valid_numbers(char* sudokus_arr, unsigned long long cell, unsigned long long top_left_sudoku_cell, int* masks) {
	unsigned long long board_offset = cell % DIMENSION;
	unsigned long long row = cell - board_offset;
	unsigned long long column = top_left_sudoku_cell + board_offset;
	unsigned long long subsquare_top_left_cell = cell - (cell % SUBSQUARE_DIMENSION) - (((cell - top_left_sudoku_cell) / DIMENSION) % SUBSQUARE_DIMENSION)*DIMENSION;
	return (row_used_numbers(sudokus_arr, row, masks) | column_used_numbers(sudokus_arr, column, masks) | subsquare_used_numbers(sudokus_arr, subsquare_top_left_cell, masks));
}


__device__
void copy_sudoku(char* src_sudoku, char* dest_sudoku) {
	for (int i = 0; i < DIMENSION*DIMENSION; i++) {
		*(dest_sudoku + i) = *(src_sudoku + i);
	}
}
__device__ void fill_masks(int* masks) {
	*masks = 0;
	for (int i = 0; i < DIMENSION; i++) {
		*(masks + i + 1) = 2 << i;
	}
}

__global__
void generate_next_permutations(char* sudokus_arr, char* sudoku_arr_new_permutations, unsigned long long* number_of_old_permutations, unsigned long long* number_of_permutations, int empty_cell, unsigned long long max_permutations, bool* max_permutations_overflow){

	
	unsigned long long tIdx = threadIdx.x + blockDim.x*blockIdx.x;
	unsigned long long id = 0;
	unsigned long long new_permutation_idx = 0;
	unsigned long long top_left_cell = 0;
	__shared__ int masks[DIMENSION + 1];
	__shared__ int valid_numbers[BLOCK_DIMENSION];
	
	fill_masks(masks);


	while (tIdx < *number_of_old_permutations && !*max_permutations_overflow) {
		
		top_left_cell = tIdx * DIMENSION * DIMENSION;
		
		valid_numbers[threadIdx.x] = get_valid_numbers(sudokus_arr, top_left_cell + empty_cell, top_left_cell, masks);
		
		
		for (int i = 1; i < DIMENSION + 1; i++) {
			if ((masks[i] & (~valid_numbers[threadIdx.x])) != 0) {
				id = atomicAdd(number_of_permutations, 1);
				new_permutation_idx = id * DIMENSION * DIMENSION;
				if (id < max_permutations) {
						copy_sudoku(sudokus_arr + top_left_cell, sudoku_arr_new_permutations + new_permutation_idx);
						*(sudoku_arr_new_permutations + new_permutation_idx + empty_cell) = (char)i;
				}
				else {
					*max_permutations_overflow = true;
					return;
				}
			}
		}
		tIdx += blockDim.x * gridDim.x;
	}		
}

__host__ __device__ void printT(char* sudoku) {
	printf("-----------------------------------------------------------------------\n");
	for (int i = 0; i < DIMENSION*DIMENSION; i++) {
		printf("%d ", sudoku[i]);
		if (i%DIMENSION == DIMENSION - 1)
			printf("\n");
	}
	printf("-----------------------------------------------------------------------\n");
}

__global__ void backtrackigKernel(char* sudokus_arr, int number_of_permutations, unsigned long long* current_sudoku_index, int* empty_cells, int empty_cells_count, bool* solved)
{
	__shared__ int masks[DIMENSION + 1];
	__shared__ int valid_numbers[BLOCK_DIMENSION];
	__shared__ int empty_cells_offsets[BLOCK_DIMENSION];
	fill_masks(masks);
	int sudoku_index = atomicAdd(current_sudoku_index, 1);
	unsigned long long sudokus_arr_idx = sudoku_index * DIMENSION * DIMENSION;
	while ((sudoku_index < number_of_permutations) && !(*solved)) {
		
		valid_numbers[threadIdx.x] = get_valid_numbers(sudokus_arr, sudokus_arr_idx + *(empty_cells + empty_cells_offsets[threadIdx.x]), sudokus_arr_idx, masks);
		
		empty_cells_offsets[threadIdx.x] = 0;
		
		for (int i = 1; i < DIMENSION + 1; i++) {
			if (!(*solved)) {
				if ((masks[i] & (~valid_numbers[threadIdx.x])) != 0) {
					sudokus_arr[sudokus_arr_idx + *(empty_cells + empty_cells_offsets[threadIdx.x])] = (char)i;
					empty_cells_offsets[threadIdx.x] += 1;
					if (empty_cells_offsets[threadIdx.x] < empty_cells_count) {
						i = 0;
						valid_numbers[threadIdx.x] = get_valid_numbers(sudokus_arr, sudokus_arr_idx + *(empty_cells + empty_cells_offsets[threadIdx.x]), sudokus_arr_idx, masks);
					}
					else {
						*solved = true;
						copy_sudoku(sudokus_arr + sudokus_arr_idx, sudokus_arr);
						printT(sudokus_arr);
						return;
					}
				}
				else {
					while (i == DIMENSION) {
						sudokus_arr[sudokus_arr_idx + *(empty_cells + empty_cells_offsets[threadIdx.x])] = 0;
						empty_cells_offsets[threadIdx.x] -= 1;
						i = sudokus_arr[sudokus_arr_idx + *(empty_cells + empty_cells_offsets[threadIdx.x])];
						sudokus_arr[sudokus_arr_idx + *(empty_cells + empty_cells_offsets[threadIdx.x])] = 0;
					}
					if (empty_cells_offsets[threadIdx.x] > -1) {
						valid_numbers[threadIdx.x] = get_valid_numbers(sudokus_arr, sudokus_arr_idx + *(empty_cells + empty_cells_offsets[threadIdx.x]), sudokus_arr_idx, masks);
					}
					else {
						sudoku_index = atomicAdd(current_sudoku_index, 1);
						break;
					}
				}
			}
			else {
				return;
			}
		}
	}
}

__host__
int get_empty_indices(char* sudoku, int* empty) {
	int count = 0;
	for (int i = 0; i < DIMENSION*DIMENSION; i++) {
		if ((int)sudoku[i] == 0) {
			empty[count++] = i;
		}
	}
	return count;
}

__host__ 
size_t get_free_memory_size() {
	size_t free_memory;
	hipMemGetInfo(&free_memory, nullptr);
	return free_memory;
}

__host__ 
unsigned long long calculate_max_number_of_permutations() {
	return (unsigned long long)(MAX_MEMORY_USAGE * (get_free_memory_size() / (2 * DIMENSION*DIMENSION * sizeof(char))));
}


__host__ 
void free_memory_GPU(char* sudokus_arr1, char* sudokus_arr2, unsigned long long* number_of_permutations1, unsigned long long* number_of_permutations2, int* empty_cells) {
	hipFree(sudokus_arr1);
	hipFree(sudokus_arr2);
	hipFree(number_of_permutations1);
	hipFree(number_of_permutations2);
	hipFree(empty_cells);
}

__host__
void err() {
	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		printf("CUDA error: %s\n", hipGetErrorString(error));
	}	
}
 
__host__
char* solve_sudokuGPU(char* sudoku) {
	
	int empty_indices[DIMENSION*DIMENSION];
	int empty_count = get_empty_indices(sudoku, empty_indices);
	unsigned long long max_permutations_number;
	char* result = (char*)malloc(sizeof(char) * DIMENSION * DIMENSION);
	
	char *sudokus_arr1, *sudokus_arr2, *max_permutations_array;
	int *empty_indicesGPU;
	unsigned long long *number_of_permutations1, *number_of_permutations2, *backtracking_sudoku_index, generated_permutations;
	bool* doWorkGPU, *doWork;
	
	hipMalloc(&number_of_permutations1, sizeof(unsigned long long));
	hipMalloc(&number_of_permutations2, sizeof(unsigned long long));
	hipMalloc(&empty_indicesGPU, sizeof(int)*empty_count);
	hipHostAlloc(&doWork, sizeof(bool), hipHostMallocMapped);
	hipHostGetDevicePointer(&doWorkGPU, doWork, 0);
	
	max_permutations_number = calculate_max_number_of_permutations();
	
	hipMalloc(&sudokus_arr1, sizeof(char)*max_permutations_number*DIMENSION*DIMENSION);
	hipMalloc(&sudokus_arr2, sizeof(char)*max_permutations_number*DIMENSION*DIMENSION);

	hipMemcpy(sudokus_arr1, sudoku, DIMENSION * DIMENSION * sizeof(char), hipMemcpyHostToDevice);
	hipMemset(number_of_permutations1, 1, 1);
	hipMemset(number_of_permutations2, 0, sizeof(int));
	hipMemcpy(empty_indicesGPU, empty_indices, empty_count * sizeof(int), hipMemcpyHostToDevice);
	
	*doWork = false;

	for (int i = 0; i < empty_count; i++) {
		if (!(i % 2)) {
			generate_next_permutations << <GRID_DIMENSION, BLOCK_DIMENSION >> > (sudokus_arr1, sudokus_arr2, number_of_permutations1, number_of_permutations2, empty_indices[i], max_permutations_number, doWorkGPU);
			hipDeviceSynchronize();
			hipMemcpy(&generated_permutations, number_of_permutations1, sizeof(unsigned long long), hipMemcpyDeviceToHost);
			hipMemset(number_of_permutations1, 0, sizeof(unsigned long long));
		}
		else {
			generate_next_permutations << <GRID_DIMENSION, BLOCK_DIMENSION >> > (sudokus_arr2, sudokus_arr1, number_of_permutations2, number_of_permutations1, empty_indices[i], max_permutations_number, doWorkGPU);
			hipDeviceSynchronize();
			hipMemcpy(&generated_permutations, number_of_permutations2, sizeof(unsigned long long), hipMemcpyDeviceToHost);
			hipMemset(number_of_permutations2, 0, sizeof(unsigned long long));
		}		
		
		if (*doWork) {
			
			if (!(i % 2)) {
				max_permutations_array = sudokus_arr1;
			}
			else {
				max_permutations_array = sudokus_arr2;
			}
	
			backtracking_sudoku_index = number_of_permutations1;
			hipMemset(backtracking_sudoku_index, 0, sizeof(unsigned long long));
			*doWork = false;
			hipMemcpy(empty_indicesGPU, &empty_indices[i], sizeof(int)*(empty_count - i), hipMemcpyHostToDevice);
			backtrackigKernel << <GRID_DIMENSION, BLOCK_DIMENSION >> > (max_permutations_array, generated_permutations, backtracking_sudoku_index, empty_indicesGPU, empty_count - i, doWork);
			hipDeviceSynchronize();
			hipMemcpy(result, max_permutations_array, sizeof(char)*DIMENSION*DIMENSION, hipMemcpyDeviceToHost);
			free_memory_GPU(sudokus_arr1, sudokus_arr2, number_of_permutations1, number_of_permutations2, empty_indicesGPU);
			return result;
		}
	}
	if (empty_count % 2) {
		hipMemcpy(result, sudokus_arr2, sizeof(char)*DIMENSION*DIMENSION, hipMemcpyDeviceToHost);
	}
	else {
		hipMemcpy(result, sudokus_arr1, sizeof(char)*DIMENSION*DIMENSION, hipMemcpyDeviceToHost);
	}
	free_memory_GPU(sudokus_arr1, sudokus_arr2, number_of_permutations1, number_of_permutations2, empty_indicesGPU);
	return result;
}

