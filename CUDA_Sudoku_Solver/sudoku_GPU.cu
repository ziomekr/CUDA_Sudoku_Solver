#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "stdio.h"
#include "stdlib.h"
#include "sudoku_GPU.cuh"


#define DIMENSION 9
#define SUBSQUARE_DIMENSION 3
#define BLOCK_DIMENSION 256
#define GRID_DIMENSION 1024*5
#define MAX_SUDOKUS_COUNT 5000000


__device__
int row_used_numbers(char* sudokus_arr, int row, int* masks) {
	int used_numbers = 0;
	for (int i = row; i < row + DIMENSION; i++) {
		used_numbers = used_numbers | masks[sudokus_arr[i]];
	}
	return used_numbers;
}

__device__
int column_used_numbers(char* sudokus_arr, int column, int* masks) {
	int used_numbers = 0;
	for (int i = 0; i < DIMENSION; i++) {
		used_numbers = used_numbers | masks[sudokus_arr[column + i * DIMENSION]];
	}
	return used_numbers;
}

__device__
int subsquare_used_numbers(char* sudokus_arr, int subsquare_top_left_cell, int* masks) {
	int used_numbers = 0;
	for (int i = subsquare_top_left_cell; i < subsquare_top_left_cell + SUBSQUARE_DIMENSION; i++) {
		for (int j = 0; j < SUBSQUARE_DIMENSION; j++) {
			used_numbers = used_numbers | masks[sudokus_arr[i + j * DIMENSION]];
		}
	}
	return used_numbers;
}

__device__
int get_valid_numbers(char* sudokus_arr, int cell, int top_left_sudoku_cell, int* masks) {
	int board_offset = cell % DIMENSION;
	int row = cell - board_offset;
	int column = top_left_sudoku_cell + board_offset;
	int subsquare_top_left_cell = cell - (cell % SUBSQUARE_DIMENSION) - (((cell - top_left_sudoku_cell) / DIMENSION) % SUBSQUARE_DIMENSION)*DIMENSION;
	return (row_used_numbers(sudokus_arr, row, masks) | column_used_numbers(sudokus_arr, column, masks) | subsquare_used_numbers(sudokus_arr, subsquare_top_left_cell, masks));
}


__device__
void copy_sudoku(char* src_sudoku, char* dest_sudoku) {
	for (int i = 0; i < DIMENSION*DIMENSION; i++) {
		*(dest_sudoku + i) = *(src_sudoku + i);
	}
}
__device__ void fill_masks(int* masks) {
	*masks = 0;
	for (int i = 0; i < DIMENSION; i++) {
		*(masks + i + 1) = 2 << i;
	}
}

__global__
void generate_next_permutations(char* sudokus_arr, char* sudoku_arr_new_permutations, int* number_of_old_permutations, int* number_of_permutations, int empty_cell){

	int tIdx = threadIdx.x + blockDim.x*blockIdx.x;
	__shared__ int masks[DIMENSION + 1];
	__shared__ int valid_numbers[BLOCK_DIMENSION];
	
	fill_masks(masks);


	while (tIdx < *number_of_old_permutations) {
		
		int top_left_cell = tIdx * DIMENSION * DIMENSION;
		valid_numbers[threadIdx.x] = get_valid_numbers(sudokus_arr, top_left_cell + empty_cell, top_left_cell , masks);
	
		
		for (int i = 1; i < DIMENSION + 1; i++) {
			if ((masks[i] & (~valid_numbers[threadIdx.x])) != 0) {
				int id = atomicAdd(number_of_permutations, 1);
				copy_sudoku(sudokus_arr + tIdx * DIMENSION*DIMENSION, sudoku_arr_new_permutations + id * DIMENSION*DIMENSION);
				sudoku_arr_new_permutations[id*DIMENSION*DIMENSION + empty_cell] = (char)i;
			}
		}
		tIdx += blockDim.x * gridDim.x;

	}
	
	
}

__device__ void printT(char* sudoku) {
	printf("-----------------------------------------------------------------------\n");
	for (int i = 0; i < DIMENSION*DIMENSION; i++) {
		printf("%d ", sudoku[i]);
		if (i%DIMENSION == DIMENSION - 1)
			printf("\n");
	}
	printf("-----------------------------------------------------------------------\n");
}

__global__ void backtrackigKernel(char* sudokus_arr, int number_of_permutations, int* current_sudoku_index, int* empty_cells, int empty_cells_count, bool solved)
{
	__shared__ int masks[DIMENSION + 1];
	__shared__ int valid_numbers[BLOCK_DIMENSION];
	__shared__ int empty_cells_offsets[BLOCK_DIMENSION];
	fill_masks(masks);
	int sudoku_index = atomicAdd(current_sudoku_index, 1);
	int sudokus_arr_idx = sudoku_index * DIMENSION * DIMENSION;
	while ((sudoku_index < number_of_permutations) && !solved) {
		
		valid_numbers[threadIdx.x] = get_valid_numbers(sudokus_arr, sudokus_arr_idx + *(empty_cells + empty_cells_offsets[threadIdx.x]), sudokus_arr_idx, masks);
		
		empty_cells_offsets[threadIdx.x] = 0;
		
		for (int i = 1; i < DIMENSION + 1; i++) {
		
			if ((masks[i] & (~valid_numbers[threadIdx.x])) != 0) {
				
				sudokus_arr[sudokus_arr_idx + *(empty_cells + empty_cells_offsets[threadIdx.x])] = (char)i;
				empty_cells_offsets[threadIdx.x] += 1;
				
				if (empty_cells_offsets[threadIdx.x] < empty_cells_count) {					
					i = 0;					
					valid_numbers[threadIdx.x] = get_valid_numbers(sudokus_arr, sudokus_arr_idx + *(empty_cells + empty_cells_offsets[threadIdx.x]), sudokus_arr_idx, masks);
				}
				else {
					solved = true;
					break;
				}
			}
			else {
				while (i == DIMENSION) {
					sudokus_arr[sudokus_arr_idx + *(empty_cells + empty_cells_offsets[threadIdx.x])] = 0;
					empty_cells_offsets[threadIdx.x] -= 1;
					i = sudokus_arr[sudokus_arr_idx + *(empty_cells + empty_cells_offsets[threadIdx.x])];
					sudokus_arr[sudokus_arr_idx + *(empty_cells + empty_cells_offsets[threadIdx.x])] = 0;
				}
				if (empty_cells_offsets[threadIdx.x] > -1) {

					valid_numbers[threadIdx.x] = get_valid_numbers(sudokus_arr, sudokus_arr_idx + *(empty_cells + empty_cells_offsets[threadIdx.x]), sudokus_arr_idx, masks);
				}
				else {
					sudoku_index = atomicAdd(current_sudoku_index, 1);
					break;
				}
			}
		}	
	}
}


void err() {
	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
	}	
}

int get_empty_indices(char* sudoku, int* empty) {
	int count = 0;
	for (int i = 0; i < DIMENSION*DIMENSION; i++) {
		if ((int)sudoku[i] == 0) {
			empty[count++] = i;
		}
	}
	return count;
}
char* solve_sudokuGPU(char* sudoku) {
//	hipEvent_t start, stop;
//	hipEventCreate(&start);
//	hipEventCreate(&stop);
//	
//	int count;
//	int empty[DIMENSION*DIMENSION];
//	int *old_perm, *new_perm;
//	char *n, *p;
//	char *result;
//	result = (char*)malloc(sizeof(char) * 81);
//	int np = 1;
//	hipEventRecord(start);
//	hipMalloc((void**)&n, MAX_SUDOKUS_COUNT * DIMENSION * DIMENSION * sizeof(char));
//	err();
//	hipMalloc((void**)&p, MAX_SUDOKUS_COUNT * DIMENSION * DIMENSION * sizeof(char));
//	err();
//
//	hipMalloc((void**)&old_perm, sizeof(int));
//	
//	hipMalloc((void**)&new_perm, sizeof(int));
//	hipMemcpy(n, sudoku, DIMENSION * DIMENSION * sizeof(char), hipMemcpyHostToDevice);
//	hipMemcpy(new_perm, &np, sizeof(int), hipMemcpyHostToDevice);
//
//	count = get_empty_indices(sudoku, empty);
//;
//	for (int i = 0; i < count; i++) {
//
//
//		hipMemcpyAsync(old_perm, new_perm, sizeof(int), hipMemcpyDeviceToDevice);
//
//		hipMemset(new_perm, 0, sizeof(int));
//
//		if (~i % 2) {
//			generate_next_permutations << <GRID_DIMENSION, BLOCK_DIMENSION >> > (n, p, old_perm, new_perm, empty[i]);
//		}
//		else {
//			generate_next_permutations << <GRID_DIMENSION, BLOCK_DIMENSION >> > (p, n, old_perm, new_perm, empty[i]);
//		}
//		err();
//	}
//	if (count % 2) {
//		hipMemcpy(result, p, 81 * sizeof(char), hipMemcpyDeviceToHost);
//	}
//	else {
//		hipMemcpy(result, n, 81 * sizeof(char), hipMemcpyDeviceToHost);
//	}
//	hipEventRecord(stop);
//	hipEventSynchronize(stop);
//
//	float milliseconds = 0;
//	hipEventElapsedTime(&milliseconds, start, stop);
//	printf("GPU time used: %f ms\n", milliseconds);
//	hipFree(n);
//	hipFree(p);
//
//	hipFree(old_perm);
//	hipFree(new_perm);

	int count = 0;
	int empty[DIMENSION*DIMENSION];
	char *n;
	int* empty_c;
	count = get_empty_indices(sudoku, empty);
	char* result;
	result = (char*)malloc(sizeof(char) * 81);
	int* c_i;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	hipMalloc((void**)&n, DIMENSION * DIMENSION * sizeof(char)*10);
	hipMalloc((void**)&empty_c, count * sizeof(int));
	hipMalloc((void**)&c_i, sizeof(int));
	
	hipMemcpy(n, sudoku, DIMENSION * DIMENSION * sizeof(char)*10, hipMemcpyHostToDevice);
	hipMemcpy(empty_c, empty, count * sizeof(int), hipMemcpyHostToDevice);
	hipMemset(c_i, 0, sizeof(int));
	backtrackigKernel << < GRID_DIMENSION, BLOCK_DIMENSION >> > (n, 10, c_i, empty_c, count, false);
	hipDeviceSynchronize();
	err();
	err();
	hipMemcpy(result, n, 81 * sizeof(char), hipMemcpyDeviceToHost);
	err();
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("GPU time used: %f ms\n", milliseconds);
	hipFree(n);
	hipFree(empty_c);
	hipFree(c_i);
	return result;

}


